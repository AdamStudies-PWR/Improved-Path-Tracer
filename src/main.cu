#include <functional>
#include <iostream>
#include <memory>

#include "containers/Vec3.hpp"
#include "renderer/RenderController.hpp"
#include "scene/SceneData.hpp"
#include "utils/CudaUtils.hpp"
#include "utils/Image.hpp"
#include "utils/Measurements.hpp"
#include "utils/InputParser.hpp"


namespace
{
using namespace tracer::containers;
using namespace tracer::renderer;
using namespace tracer::scene;
using namespace tracer::utils;
}  // namespace

int main(int argc, char* argv[])
{
    if (not checkCudaSupport())
    {
        return 0;
    }

    InputParser inputParser((argc - 1), argv);
    if (not inputParser.isInputValid())
    {
        return 0;
    }

    SceneData sceneData(inputParser.getScenePath());
    if (not sceneData.initScene())
    {
        return 0;
    }

    std::ostringstream filename;
    filename << inputParser.getSceneName() << "D" << +inputParser.getMaxDepth() << "S"
        << +inputParser.getSamplingRate();

    auto controller = std::make_shared<RenderController>(sceneData, inputParser.getSamplingRate(),
        inputParser.getMaxDepth());
    const auto wrappedRender = [controller, sceneData]() -> const std::vector<Vec3> {
        return controller->start(sceneData.getObjectsData());
    };
    const auto image = measure(filename.str(), std::move(wrappedRender));

    hipError_t maybeError = hipGetLastError();
    if (maybeError != hipSuccess)
    {
        return 1;
    }

    saveImage(image, sceneData.getHeight(), sceneData.getWidth(), filename.str());

    return 0;
}
