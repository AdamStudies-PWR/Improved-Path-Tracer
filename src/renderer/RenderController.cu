#include "hip/hip_runtime.h"
#include "renderer/RenderController.hpp"

#include <iostream>

#include "objects/AObject.hpp"

#include "Constants.hpp"
#include "Renderer.cu"
#include "ImageData.hpp"


namespace tracer::renderer
{

namespace
{
using namespace containers;
using namespace scene;
using namespace scene::objects;

void cudaErrorCheck(const std::string& message)
{
    hipError_t maybeError = hipGetLastError();
    if (maybeError != hipSuccess)
    {
        std::cout << message << " - error: " << hipGetErrorString(maybeError) << std::endl;
    }
}

__global__ void cudaCreateObjects(AObject** objects, ObjectData* objectsData)
{
    if (objectsData[threadIdx.x].objectType_ == SphereData)
    {
        objects[threadIdx.x] = new Sphere(objectsData[threadIdx.x].radius_, objectsData[threadIdx.x].position_,
            objectsData[threadIdx.x].emission_, objectsData[threadIdx.x].color_,
            objectsData[threadIdx.x].reflectionType_);
    }
    else if (objectsData[threadIdx.x].objectType_ == PlaneData)
    {
        objects[threadIdx.x] = new Plane(objectsData[threadIdx.x].north_, objectsData[threadIdx.x].east_,
            objectsData[threadIdx.x].position_, objectsData[threadIdx.x].emission_, objectsData[threadIdx.x].color_,
            objectsData[threadIdx.x].reflectionType_);
    }
}
}  // namespace

RenderContoller::RenderContoller(SceneData& sceneData, const uint32_t samples, const uint8_t maxDepth)
    : sceneData_(sceneData)
    , maxDepth_(maxDepth)
    , samples_(samples)
{}

std::vector<containers::Vec3> RenderContoller::start()
{
    auto objectDataVec = sceneData_.getObjectsData();

    AObject** devObjects;
    hipMalloc((void**)&devObjects, sizeof(AObject) * objectDataVec.size());
    cudaErrorCheck("Copy object blueprint data");

    ObjectData* devData;
    hipMalloc((void**)&devData, sizeof(ObjectData) * objectDataVec.size());
    hipMemcpy(devData, objectDataVec.data(), sizeof(ObjectData) * objectDataVec.size(), hipMemcpyHostToDevice);
    cudaErrorCheck("Allocate memory for objects");

    cudaCreateObjects <<<1, objectDataVec.size()>>> (devObjects, devData);
    cudaErrorCheck("cudaCreateObjects kernel");

    hipFree(devData);
    cudaErrorCheck("Clear object blueprint data");

    auto camera = sceneData_.getCamera();
    Camera* devCamera;
    hipMalloc((void**)&devCamera, sizeof(Camera));
    hipMemcpy(devCamera, &camera, sizeof(Camera), hipMemcpyHostToDevice);
    cudaErrorCheck("Copy camera data");

    Vec3* devVecZ;
    hipMalloc((void**)&devVecZ, sizeof(Vec3));
    hipMemcpy(devVecZ, &(camera.direction_%camera.orientation_).norm(), sizeof(Vec3), hipMemcpyHostToDevice);
    cudaErrorCheck("Copy vecZ data");

    ImageData* devImageData;
    const auto imageProperties = ImageData(sceneData_.getWidth(), sceneData_.getHeight(), samples_, maxDepth_,
        objectDataVec.size());
    hipMalloc((void**)&devImageData, sizeof(ImageData));
    hipMemcpy(devImageData, &imageProperties, sizeof(ImageData), hipMemcpyHostToDevice);
    cudaErrorCheck("Copy image properties data");

    const auto imageSize = sceneData_.getHeight() * sceneData_.getWidth() * sizeof(Vec3);
    Vec3* devImage;
    hipMalloc((void**)&devImage, imageSize);
    hipMemset(devImage, 0, imageSize);
    cudaErrorCheck("Set image array");

    const auto numThreads = (sceneData_.getWidth() <= BLOCK_SIZE) ? sceneData_.getWidth() : BLOCK_SIZE;
    const auto numBlocks = (sceneData_.getHeight() <= BLOCK_SIZE) ? sceneData_.getHeight() : BLOCK_SIZE;
    cudaMain <<<numBlocks, numThreads>>> (devImage, devObjects, devCamera, devVecZ, devImageData);
    cudaErrorCheck("cudaMain kernel");

    Vec3* imagePtr = (Vec3*)malloc(imageSize);
    hipMemcpy(imagePtr, devImage, imageSize, hipMemcpyDeviceToHost);
    cudaErrorCheck("Copy to device");

    hipDeviceReset();
    cudaErrorCheck("Reset device");

    const auto image = convertToVector(imagePtr);
    free(imagePtr);

    return image;
}

std::vector<Vec3> RenderContoller::convertToVector(Vec3* imagePtr)
{
    std::vector<Vec3> image;
    for (uint32_t iter = 0; iter < sceneData_.getHeight() * sceneData_.getWidth(); iter++)
    {
        image.push_back(imagePtr[iter]);
    }

    return image;
}

}  // namespace tracer::renderer
