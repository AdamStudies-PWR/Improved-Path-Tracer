#include "hip/hip_runtime.h"
#include "renderer/RenderController.hpp"

#include <iostream>

#include "objects/AObject.hpp"

#include "Constants.hpp"
#include "helpers/SceneConstants.hpp"
#include "helpers/PixelData.hpp"
#include "RendererCPU.hpp"
#include "RendererGPU.cu"


namespace tracer::renderer
{

namespace
{
using namespace containers;
using namespace scene;
using namespace scene::objects;

const float FOV_SCALE = 0.0009;

void cudaErrorCheck(const std::string& message)
{
    hipError_t maybeError = hipGetLastError();
    if (maybeError != hipSuccess)
    {
        std::cout << message << " - error: " << hipGetErrorString(maybeError) << std::endl;
    }
}

__global__ void cudaCreateObjects(AObject** objects, ObjectData* objectsData)
{
    if (objectsData[threadIdx.x].objectType_ == SphereData)
    {
        objects[threadIdx.x] = new Sphere(objectsData[threadIdx.x].radius_, objectsData[threadIdx.x].position_,
            objectsData[threadIdx.x].emission_, objectsData[threadIdx.x].color_,
            objectsData[threadIdx.x].reflectionType_);
    }
    else if (objectsData[threadIdx.x].objectType_ == PlaneData)
    {
        objects[threadIdx.x] = new Plane(objectsData[threadIdx.x].north_, objectsData[threadIdx.x].east_,
            objectsData[threadIdx.x].position_, objectsData[threadIdx.x].emission_, objectsData[threadIdx.x].color_,
            objectsData[threadIdx.x].reflectionType_);
    }
}

std::vector<uint32_t> prepareRandomSeends(const uint32_t ammount)
{
    std::vector<uint32_t> seeds;
    for (uint32_t i=0; i<ammount; i++)
    {
        seeds.push_back(rand());
    }

    return seeds;
}

}  // namespace

RenderController::RenderController(SceneData& sceneData, const uint32_t samples, const uint8_t maxDepth)
    : sceneData_(sceneData)
    , maxDepth_(maxDepth)
    , samples_(samples)
    , image_(std::vector<Vec3> (sceneData_.getWidth() * sceneData_.getHeight()))
{}

std::vector<containers::Vec3> RenderController::start()
{
    auto objectDataVec = sceneData_.getObjectsData();

    AObject** devObjects;
    hipMalloc((void**)&devObjects, sizeof(AObject) * objectDataVec.size());
    cudaErrorCheck("Copy object blueprint data");

    ObjectData* devData;
    hipMalloc((void**)&devData, sizeof(ObjectData) * objectDataVec.size());
    hipMemcpy(devData, objectDataVec.data(), sizeof(ObjectData) * objectDataVec.size(), hipMemcpyHostToDevice);
    cudaErrorCheck("Allocate memory for objects");

    cudaCreateObjects <<<1, objectDataVec.size()>>> (devObjects, devData);
    cudaErrorCheck("cudaCreateObjects kernel");

    hipFree(devData);
    cudaErrorCheck("Clear object blueprint data");

    auto camera = sceneData_.getCamera();
    const Vec3 vecZ = (camera.direction_%camera.orientation_).norm();

    SceneConstants* devConstants;
    const auto constants = SceneConstants(camera.orientation_, vecZ, camera.origin_, camera.direction_, samples_,
        maxDepth_, objectDataVec.size());
    hipMalloc((void**)&devConstants, sizeof(SceneConstants));
    hipMemcpy(devConstants, &constants, sizeof(SceneConstants), hipMemcpyHostToDevice);
    cudaErrorCheck("Copy scene constants");

    const auto callback = [this, devObjects, devConstants, &vecZ](uint32_t z)
    {
        renderGPU(z, devObjects, devConstants, vecZ);
    };

    renderCPU(sceneData_.getHeight(), callback);

    hipDeviceReset();
    cudaErrorCheck("Reset device");

    return image_;
}

void RenderController::renderGPU(const uint32_t z, AObject** devObjects, SceneConstants* devConstants, const Vec3& vecZ)
{
    Vec3* devSamples;
    hipMalloc((void**)&devSamples, sizeof(Vec3) * samples_);
    cudaErrorCheck("Set image array");

    PixelData* devPixelData;
    hipMalloc((void**)&devPixelData, sizeof(PixelData));
    cudaErrorCheck("Allocate pixel data");

    for (uint32_t x=0; x<sceneData_.getWidth(); x++)
    {
        const auto index = z * sceneData_.getWidth() + x;
        image_[index] = startKernel(devObjects, devSamples, devPixelData, devConstants, vecZ, x, z);
    }

    hipFree(devSamples);
    hipFree(devPixelData);
    cudaErrorCheck("Free samples and pixel data arrays");
}

Vec3 RenderController::startKernel(AObject** devObjects, Vec3* devSamples, PixelData* devPixelData,
    SceneConstants* devConstants, const Vec3& vecZ, const uint32_t pixelX, const uint32_t pixelZ)
{
    const auto vecX = sceneData_.getCamera().orientation_;
    const auto center = sceneData_.getCamera().origin_;
    const auto direction = sceneData_.getCamera().direction_;

    auto correctionX = (sceneData_.getWidth() % 2 == 0) ? 0.5 : 0.0;
    auto correctionZ = (sceneData_.getHeight() % 2 == 0) ? 0.5 : 0.0;
    double stepX = (pixelX < sceneData_.getWidth()/2)
        ? sceneData_.getWidth()/2 - pixelX - correctionX
        : ((double)sceneData_.getWidth()/2 - pixelX - 1.0) + ((correctionX == 0.0) ? 1.0 : correctionX);
    double stepZ = (pixelZ < sceneData_.getHeight()/2)
        ? sceneData_.getHeight()/2 - pixelZ - correctionZ
        : ((double)sceneData_.getHeight()/2 - pixelZ - 1.0) + ((correctionZ == 0.0) ? 1.0 : correctionZ);

    const auto gaze = (direction + vecX*stepX*FOV_SCALE + vecZ*stepZ*FOV_SCALE).norm();

    const auto pixelData = PixelData(stepX, stepZ, gaze);

    hipMemcpy(devPixelData, &pixelData, sizeof(PixelData), hipMemcpyHostToDevice);
    cudaErrorCheck("Allocate and copy pixel data");

    hipMemset(devSamples, 0, sizeof(Vec3) * samples_);
    cudaErrorCheck("Zero sample array");

    const auto numThreads = (samples_ <= THREAD_LIMIT) ? samples_ : THREAD_LIMIT;

    uint32_t* devSeeds;
    hipMalloc((void**)&devSeeds, sizeof(uint32_t) * numThreads);
    hipMemcpy(devSeeds, prepareRandomSeends(numThreads).data(), sizeof(uint32_t) * numThreads,
        hipMemcpyHostToDevice);
    cudaErrorCheck("Prepare random seeds");

    cudaMain <<<1, numThreads>>> (devSamples, devObjects, devConstants, devPixelData, devSeeds);
    cudaErrorCheck("Main kernel");

    Vec3* samplesPtr = (Vec3*)malloc(sizeof(Vec3) * samples_);
    hipMemcpy(samplesPtr, devSamples, sizeof(Vec3) * samples_, hipMemcpyDeviceToHost);
    cudaErrorCheck("Copy samples from device");

    auto pixel = Vec3();
    for (uint32_t i=0; i<samples_; i++)
    {
        pixel = pixel + samplesPtr[i];
    }

    free(samplesPtr);
    hipFree(devSeeds);

    pixel.xx_ = pixel.xx_/samples_;
    pixel.yy_ = pixel.yy_/samples_;
    pixel.zz_ = pixel.zz_/samples_;

    return pixel;
}

}  // namespace tracer::renderer
