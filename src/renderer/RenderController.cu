#include "hip/hip_runtime.h"
#include "renderer/RenderController.hpp"

#include <iostream>

#include "objects/AObject.hpp"

#include "Constants.hpp"
#include "Renderer.cu"


namespace tracer::renderer
{

namespace
{
using namespace containers;
using namespace scene;
using namespace scene::objects;

__global__ void cudaCreateObjects(AObject** objects, ObjectData* objectsData)
{
    if (objectsData[threadIdx.x].objectType_ == SphereData)
    {
        objects[threadIdx.x] = new Sphere(objectsData[threadIdx.x].radius_, objectsData[threadIdx.x].position_,
            objectsData[threadIdx.x].emission_, objectsData[threadIdx.x].color_,
            objectsData[threadIdx.x].reflectionType_);
    }
    else if (objectsData[threadIdx.x].objectType_ == PlaneData)
    {
        objects[threadIdx.x] = new Plane(objectsData[threadIdx.x].north_, objectsData[threadIdx.x].east_,
            objectsData[threadIdx.x].position_, objectsData[threadIdx.x].emission_, objectsData[threadIdx.x].color_,
            objectsData[threadIdx.x].reflectionType_);
    }
}
}  // namespace

RenderContoller::RenderContoller(SceneData& sceneData, const uint32_t samples, const uint8_t maxDepth)
    : sceneData_(sceneData)
    , maxDepth_(maxDepth)
    , samples_(samples)
{}

std::vector<containers::Vec3> RenderContoller::start()
{
    auto objectDataVec = sceneData_.getObjectsData();

    AObject** devObjects;
    hipMalloc((void**)&devObjects, sizeof(AObject) * objectDataVec.size());

    ObjectData* devData;
    hipMalloc((void**)&devData, sizeof(ObjectData) * objectDataVec.size());
    hipMemcpy(devData, objectDataVec.data(), sizeof(ObjectData) * objectDataVec.size(), hipMemcpyHostToDevice);

    cudaCreateObjects <<<1, objectDataVec.size()>>> (devObjects, devData);

    hipFree(devData);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    auto camera = sceneData_.getCamera();
    const Vec3 vecZ = (camera.direction_%camera.orientation_).norm();

    const auto imageSize = sceneData_.getHeight() * sceneData_.getWidth() * sizeof(Vec3);
    Vec3* devImage;
    hipMalloc((void**)&devImage, imageSize);
    hipMemset(devImage, 0, imageSize);

    const auto numThreads = (sceneData_.getWidth() <= BLOCK_SIZE) ? sceneData_.getWidth() : BLOCK_SIZE;
    const auto numBlocks = (sceneData_.getHeight() <= BLOCK_SIZE) ? sceneData_.getHeight() : BLOCK_SIZE;
    cudaMain <<<numBlocks, numThreads>>> (devImage, devObjects, objectDataVec.size(), sceneData_.getWidth(),
        sceneData_.getHeight(), camera, vecZ, samples_, maxDepth_);

    Vec3* imagePtr = (Vec3*)malloc(imageSize);
    hipMemcpy(imagePtr, devImage, imageSize, hipMemcpyDeviceToHost);
    hipDeviceReset();

    const auto image = convertToVector(imagePtr);
    free(imagePtr);

    return image;
    // return {};
}

std::vector<Vec3> RenderContoller::convertToVector(Vec3* imagePtr)
{
    std::vector<Vec3> image;
    for (uint32_t iter = 0; iter < sceneData_.getHeight() * sceneData_.getWidth(); iter++)
    {
        image.push_back(imagePtr[iter]);
    }

    return image;
}

}  // namespace tracer::renderer
