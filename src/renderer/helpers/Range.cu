
#include <hip/hip_runtime.h>
#include <cstdint>

namespace tracer::renderer
{

struct Range
{
    __device__ Range(uint32_t startX, uint32_t startZ, uint32_t endX, uint32_t endZ)
        : startX_(startX)
        , startZ_(startZ)
        , endX_(endX)
        , endZ_(endZ)
    {}

    const uint32_t startX_;
    const uint32_t startZ_;
    const uint32_t endX_;
    const uint32_t endZ_;
};

}  // namespace tracer::renderer
