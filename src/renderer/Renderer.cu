#include "hip/hip_runtime.h"
#include <stdio.h>

#include <algorithm>

#include "containers/Vec3.hpp"
#include "objects/AObject.hpp"
#include "objects/Plane.cu"
#include "objects/Sphere.cu"
#include "scene/objects/Camera.hpp"
#include "scene/objects/ObjectData.hpp"
#include "scene/objects/RayData.hpp"
#include "utils/CudaUtils.hpp"

#include "Constants.hpp"
#include "helpers/HitData.cu"
#include "helpers/ImageData.hpp"
#include "helpers/Range.cu"
#include "helpers/RenderData.cu"


namespace tracer::renderer
{

namespace
{
using namespace containers;
using namespace scene;
using namespace scene::objects;
using namespace utils;

const float FOV_SCALE = 0.0009;
const uint16_t VIEWPORT_DISTANCE = 140;
const double INF = 1e20;

__device__ uint32_t counter = 0;

__device__ inline Range calculateRange(const uint32_t idX, const uint32_t idZ, const uint32_t width,
    const uint32_t height)
{
    auto widthAssigned = width/THREAD_SIZE;
    const auto widthOverflow = width % THREAD_SIZE;
    const auto startWidth = idX * widthAssigned + ((idX < widthOverflow) ? idX : widthOverflow);
    widthAssigned = widthAssigned + ((idX < widthOverflow) ? 1 : 0);

    auto heightAssigned = height/BLOCK_SIZE;
    const auto heightOverflow = height % BLOCK_SIZE;
    const auto startHeight = idZ * heightAssigned + ((idZ < heightOverflow) ? idZ : heightOverflow);
    heightAssigned = heightAssigned + ((idZ < heightOverflow) ? 1 : 0);

    return Range(startWidth, startHeight, (startWidth + widthAssigned), (startHeight + heightAssigned));
}

__device__ inline HitData getHitObjectAndDistance(AObject** objects, const containers::Ray& ray,
    const uint32_t objectCount)
{
    int index = -1;
    double distance = INF;

    for (uint32_t i = 0; i < objectCount; i++)
    {
        auto temp = objects[i]->intersect(ray);
        if (temp && temp < distance)
        {
            distance = temp;
            index = i;
        }
    }

    return HitData(index, distance);
}

__device__ inline Vec3 findLight(const RenderData& data, const AObject* lastObject, const Ray& ray,
    const Vec3& intersection)
{
    const bool isNormalNegative = (lastObject->getNormal(intersection, ray.direction_) < 0);

    Vec3 emission = Vec3();
    Vec3 color = Vec3();

    for (uint32_t i = 0; i < data.lightCount_; i++)
    {
        int32_t lightId = -1;
        Vec3 direction;
        Vec3 target;

        const auto extremes = data.lights_[i]->getExtremes();
        for (uint8_t j = 0; j < data.lights_[i]->getExtremesCount(); j++)
        {
            const auto distance = intersection.distance(extremes[j]);
            if (distance < INF)
            {
                target = extremes[j];
                direction = (target - intersection).norm();
                if ((lastObject->getNormal(intersection, direction * -1) < 0) == isNormalNegative)
                {
                    const auto propHitData =
                        getHitObjectAndDistance(data.props_, Ray(intersection, direction), data.propCount_);
                    if (propHitData.distance_ < distance && propHitData.distance_ > 0.0)
                    {
                        continue;
                    }

                    lightId = i;
                    break;
                }
            }
        }

        if (lightId == -1)
        {
            continue;
        }

        const auto light = data.lights_[lightId];

        const auto lightAngle = light->getAngle(target, direction);
        const auto lightFactor = lightAngle/M_PI_2;
        const auto objectAngle = lastObject->getAngle(intersection, (direction * -1));
        const auto objectFactor = objectAngle/M_PI_2;
        auto factor = lightFactor * objectFactor;
        factor = (factor < 0.0) ? factor * -1 : factor;

        color = color + light->getColor();
        emission = emission + (light->getEmission() * factor);
    }

    if (emission == Vec3())
    {
        return Vec3();
    }

    auto materialFactor = 1.0;
    if (lastObject->getReflectionType() == Refractive)
    {
        materialFactor = materialFactor * 0.08;
    }
    else if (lastObject->getReflectionType() == Refractive)
    {
        materialFactor = materialFactor * 0.4;
    }

    return (emission * materialFactor) + color.mult(Vec3());
}

__device__ inline Vec3 deepLayers(const RenderData& data, Ray ray, uint8_t depth, EReflectionType previosType)
{
    Vec3* objectEmissions = new Vec3[data.maxDepth_ - 2];
    Vec3* objectColors = new Vec3[data.maxDepth_ - 2];

    for (; depth < data.maxDepth_; depth++)
    {
        const auto propHitData = getHitObjectAndDistance(data.props_, ray, data.propCount_);
        const auto lightHitData = getHitObjectAndDistance(data.lights_, ray, data.lightCount_);
        if (propHitData.index_ == -1 && lightHitData.index_ == -1) break;

        if (lightHitData.distance_ < propHitData.distance_)
        {
            const auto light = data.lights_[lightHitData.index_];
            objectEmissions[depth - 2] = light->getEmission() * 1.5;
            objectColors[depth - 2] = light->getColor();
            depth++;
            break;
        }

        const auto& object = data.props_[propHitData.index_];
        const auto intersection = ray.origin_ + ray.direction_ * propHitData.distance_;
        Vec3 pixel = findLight(data, object, ray, intersection);
        objectEmissions[depth - 2] = pixel;
        objectColors[depth - 2] = object->getColor();
        if ((previosType == Diffuse) and (object->getReflectionType() == Diffuse))
        {
            depth++;
            break;
        }

        previosType = object->getReflectionType();
        const auto reflected = object->calculateReflections(intersection, ray.direction_, data.state_, depth);
        ray = reflected.ray_;
    }

    Vec3 pixel = Vec3();
    for (int8_t i=(depth - 3); i>= 0; i--)
    {
        pixel = objectEmissions[i] + objectColors[i].mult(pixel);
    }

    delete objectEmissions;
    delete objectColors;

    return pixel;
}

__device__ inline Vec3 secondLayer(const RenderData& data, Ray ray, uint8_t& depth, const EReflectionType& previosType,
    const bool isLight)
{
    const auto propHitData = getHitObjectAndDistance(data.props_, ray, data.propCount_);
    const auto lightHitData = getHitObjectAndDistance(data.lights_, ray, data.lightCount_);
    if (propHitData.index_ == -1 && lightHitData.index_ == -1)
    {
        return Vec3();
    }

    if (lightHitData.distance_ < propHitData.distance_)
    {
        const auto mult = (previosType != Diffuse) ? 40: 1;
        const auto light = data.lights_[lightHitData.index_];
        return light->getEmission() + light->getColor().mult(Vec3()) * mult;
    }

    const auto& object = data.props_[propHitData.index_];
    const auto intersection = ray.origin_ + ray.direction_ * propHitData.distance_;
    Vec3 pixel = findLight(data, object, ray, intersection);
    if ((previosType == Diffuse) and (object->getReflectionType() == Diffuse) and not (pixel == Vec3()))
    {
        return object->getColor().mult(pixel);
    }

    const auto reflected = object->calculateReflections(intersection, ray.direction_, data.state_, depth);
    depth++;
    auto glassBonus =
        ((object->getReflectionType() != Diffuse) and (previosType == Diffuse) and not isLight) ? 1.75 : 1;
    pixel = pixel + deepLayers(data, reflected.ray_, depth, object->getReflectionType()) * reflected.power_
        * glassBonus;
    if (reflected.useSecond_)
    {
        pixel = pixel + deepLayers(data, reflected.secondRay_, depth, object->getReflectionType())
            * reflected.secondPower_;
    }

    return object->getColor().mult(pixel);
}

__device__ inline Vec3 firstLayer(const RenderData& data, Ray ray)
{
    uint8_t depth = 0;
    const auto propHitData = getHitObjectAndDistance(data.props_, ray, data.propCount_);
    const auto lightHitData = getHitObjectAndDistance(data.lights_, ray, data.lightCount_);
    if (propHitData.index_ == -1 && lightHitData.index_ == -1)
    {
        return Vec3();
    }

    if (lightHitData.distance_ < propHitData.distance_)
    {
        const auto light = data.lights_[lightHitData.index_];
        return (light->getEmission() + light->getColor().mult(Vec3())) * 10;
    }

    const auto& object = data.props_[propHitData.index_];
    const auto intersection = ray.origin_ + ray.direction_ * propHitData.distance_;
    const auto reflected = object->calculateReflections(intersection, ray.direction_, data.state_, depth);

    depth++;
    Vec3 pixel = findLight(data, object, ray, intersection);
    bool foundLight = (pixel != Vec3());
    pixel = pixel + secondLayer(data, reflected.ray_, depth, object->getReflectionType(), foundLight) * reflected.power_;
    if (reflected.useSecond_)
    {
        pixel = pixel + secondLayer(data, reflected.secondRay_, depth, object->getReflectionType(), foundLight)
            * reflected.secondPower_;
    }

    return object->getColor().mult(pixel);
}

__device__ inline Vec3 samplePixel(const RenderData& data, const Camera* camera, ImageData* imageProperties,
    const Vec3* vecZ, const uint32_t pixelX, const uint32_t pixelZ)
{
    const auto vecX = camera->orientation_;
    const auto width = imageProperties->width_;
    const auto height = imageProperties->height_;

    auto correctionX = (width % 2 == 0) ? 0.5 : 0.0;
    auto correctionZ = (width % 2 == 0) ? 0.5 : 0.0;
    double stepX = (pixelX < width/2)
        ? width/2 - pixelX - correctionX
        : ((double)width/2 - pixelX - 1.0) + ((correctionX == 0.0) ? 1.0 : correctionX);
    double stepZ = (pixelZ < height/2)
        ? height/2 - pixelZ - correctionZ
        : ((double)height/2 - pixelZ - 1.0) + ((correctionZ == 0.0) ? 1.0 : correctionZ);

    const auto gaze = (camera->direction_ + vecX*stepX*FOV_SCALE + (*vecZ)*stepZ*FOV_SCALE).norm();

    Vec3 pixel = Vec3();
    for (uint32_t i = 0;  i < imageProperties->samples_; i++)
    {
        // Tent filter
        const auto xFactor = tent_filter(data.state_);
        const auto zFactor = tent_filter(data.state_);
        const auto tentFilter = vecX*xFactor + (*vecZ)*zFactor;
        // Tent filter

        const auto origin = camera->origin_ + vecX*stepX + (*vecZ)*stepZ + tentFilter;
        pixel = pixel + firstLayer(data, Ray(origin + camera->direction_ * VIEWPORT_DISTANCE, gaze));
    }

    pixel.xx_ = pixel.xx_/imageProperties->samples_;
    pixel.yy_ = pixel.yy_/imageProperties->samples_;
    pixel.zz_ = pixel.zz_/imageProperties->samples_;

    return pixel;
}
}  // namespace

__global__ void cudaMain(Vec3* image, AObject** props, AObject** lights, Camera* camera, Vec3* vecZ,
    ImageData* imageProperties)
{
    if (blockIdx.x == 0 and threadIdx.x == 0)
    {
        printf("\rRendering %.2f%%", (float)counter);
    }

    hiprandState state;
    auto seed = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(123456, seed, 0, &state);

    RenderData data {props, lights, imageProperties->propCount_, imageProperties->lightCount_,
        imageProperties->maxDepth_, state};

    const auto totalPixels = imageProperties->width_ * imageProperties->height_;
    const auto range = calculateRange(threadIdx.x, blockIdx.x, imageProperties->width_, imageProperties->height_);
    for (uint32_t z = range.startZ_; z < range.endZ_; z++)
    {
        for (uint32_t x = range.startX_; x < range.endX_; x++)
        {
            const auto index = z * imageProperties->width_ + x;
            image[index] = samplePixel(data, camera, imageProperties, vecZ, x, z);
            atomicAdd(&counter, 1);
        }
        printf("\rRendering %.2f%%", ((float)counter/(totalPixels)*100));
    }
}

}  // namespace tracer::render
