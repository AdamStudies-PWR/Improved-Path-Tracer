#include "hip/hip_runtime.h"
#include <stdio.h>

#include "containers/Vec3.hpp"
#include "objects/AObject.hpp"
#include "objects/Plane.cu"
#include "objects/Sphere.cu"
#include "scene/objects/Camera.hpp"
#include "scene/objects/ObjectData.hpp"
#include "scene/objects/RayData.hpp"
#include "utils/CudaUtils.hpp"

#include "Coordinates.cu"
#include "Constants.hpp"
#include "HitData.cu"


namespace tracer::renderer
{

namespace
{
using namespace containers;
using namespace scene;
using namespace scene::objects;
using namespace utils;

const float FOV_SCALE = 0.0009;
const uint16_t VIEWPORT_DISTANCE = 140;
const double INF = 1e20;

__device__ Coordinates calculateCoordinates(const uint32_t idX, const uint32_t idZ,
    const uint32_t width, const uint32_t height)
{
    if (width <= BLOCK_SIZE && height <= BLOCK_SIZE)
    {
        return Coordinates(idX, idZ, 0, 0);
    }

    const uint32_t xAddition = width % BLOCK_SIZE;
    const uint32_t zAddition = height % BLOCK_SIZE;
    auto xStepping = width/BLOCK_SIZE;
    auto zStepping = height/BLOCK_SIZE;

    auto pixelX = idX * xStepping + ((idX > xAddition) ? xAddition : idX);
    auto pixelZ = idZ * zStepping + ((idZ > zAddition) ? zAddition : idZ);

    xStepping = xStepping + ((xAddition <= 0) ? 0 : ((idX < xAddition) ? 1 : 0));
    zStepping = zStepping + ((zAddition <= 0) ? 0 : ((idZ < zAddition) ? 1 : 0));

    return Coordinates(pixelX, pixelZ, xStepping, zStepping);
}
}  // namespace

class Renderer
{
public:
    __device__ Renderer(const uint32_t samples, const uint32_t width, const uint32_t height, const uint8_t maxDepth,
        const Camera& camera)
        : camera_(camera)
        , height_(height)
        , samples_(samples)
        , width_(width)
        , maxDepth_(maxDepth)
        , objectsCount_(0)
    {}

    __device__ void setUp(ObjectData* objectsData, const uint32_t objectCount)
    {
        objectsCount_ = objectCount;
        objects_ = new AObject*[objectsCount_];
        for (uint32_t i=0; i<objectsCount_; i++)
        {
            if (objectsData[i].objectType_ == SphereData)
            {
                objects_[i] = new Sphere(objectsData[i].radius_, objectsData[i].position_, objectsData[i].emission_,
                    objectsData[i].color_, objectsData[i].reflectionType_);
            }
            else if (objectsData[i].objectType_ == PlaneData)
            {
                objects_[i] = new Plane(objectsData[i].north_, objectsData[i].east_, objectsData[i].position_,
                    objectsData[i].emission_, objectsData[i].color_, objectsData[i].reflectionType_);
            }
        }
    }

    __device__ void start(Vec3* image, const Vec3& vecZ)
    {
        const auto coordinates = calculateCoordinates(threadIdx.x, blockIdx.x, width_, height_);
        const auto limitZ = coordinates.zz_ + coordinates.loopZ_ + 1;
        const auto limitX = coordinates.xx_ + coordinates.loopX_ + 1;

        hiprandState state;
        auto seed = threadIdx.x + blockIdx.x * blockDim.x;
        hiprand_init(123456, seed, 0, &state);

        for (uint32_t z=coordinates.zz_; z<limitZ; z++)
        {
            for (uint32_t x=coordinates.xx_; x<limitX; x++)
            {
                const auto index = z * width_ + x;
                image[index] = samplePixel(camera_.orientation_, vecZ, x, z, samples_, state);
            }
        }
    }

private:
    __device__ Vec3 samplePixel(const containers::Vec3& vecX, const containers::Vec3& vecZ, const uint32_t pixelX,
    const uint32_t pixelZ, const uint32_t samples, hiprandState& state) const
    {
        const auto center = camera_.origin_;
        const auto direction = camera_.direction_;

        auto correctionX = (width_ % 2 == 0) ? 0.5 : 0.0;
        auto correctionZ = (width_ % 2 == 0) ? 0.5 : 0.0;
        double stepX = (pixelX < width_/2)
            ? width_/2 - pixelX - correctionX
            : ((double)width_/2 - pixelX - 1.0) + ((correctionX == 0.0) ? 1.0 : correctionX);
        double stepZ = (pixelZ < height_/2)
            ? height_/2 - pixelZ - correctionZ
            : ((double)height_/2 - pixelZ - 1.0) + ((correctionZ == 0.0) ? 1.0 : correctionZ);

        const auto gaze = (direction + vecX*stepX*FOV_SCALE + vecZ*stepZ*FOV_SCALE).norm();

        Vec3 pixel = Vec3();
        for (uint32_t i=0; i<samples; i++)
        {
            // Tent filter
            const auto xFactor = tent_filter(state);
            const auto zFactor = tent_filter(state);
            const auto tentFilter = vecX * xFactor + vecZ * zFactor;
            // Tent filter

            const auto origin = center + vecX*stepX + vecZ*stepZ + tentFilter;
            pixel = pixel + firstLayer(Ray(origin + direction * VIEWPORT_DISTANCE, gaze), state);
        }

        pixel.xx_ = pixel.xx_/samples;
        pixel.yy_ = pixel.yy_/samples;
        pixel.zz_ = pixel.zz_/samples;

        return pixel;
    }

    __device__ Vec3 firstLayer(Ray ray, hiprandState& state) const
    {
        uint8_t depth = 0;
        const auto hitData = getHitObjectAndDistance(ray);
        if (hitData.index_ == -1) return Vec3();

        const auto& object = objects_[hitData.index_];
        const auto intersection = ray.origin_ + ray.direction_ * hitData.distance_;
        const auto reflected = object->calculateReflections(intersection, ray.direction_, state, depth);

        depth++;
        Vec3 backData;
        if (depth < maxDepth_)
        {
            backData = secondLayer(reflected.ray_, depth, state) * reflected.power_;
            if (reflected.useSecond_)
            {
                backData = backData + secondLayer(reflected.secondRay_, depth, state) * reflected.secondPower_;
            }
        }

        return object->getEmission() + object->getColor().mult(backData);
    }

    __device__ Vec3 secondLayer(Ray ray, uint8_t& depth, hiprandState& state) const
    {
        const auto hitData = getHitObjectAndDistance(ray);
        if (hitData.index_ == -1) return Vec3();

        const auto& object = objects_[hitData.index_];
        const auto intersection = ray.origin_ + ray.direction_ * hitData.distance_;
        const auto reflected = object->calculateReflections(intersection, ray.direction_, state, depth);

        depth++;
        Vec3 backData;
        if (depth < maxDepth_)
        {
            backData = deepLayers(reflected.ray_, depth, state) * reflected.power_;
            if (reflected.useSecond_)
            {
                backData = backData + deepLayers(reflected.secondRay_, depth, state) * reflected.secondPower_;
            }
        }

        return object->getEmission() + object->getColor().mult(backData);
    }

    __device__ Vec3 deepLayers(Ray ray, uint8_t depth, hiprandState& state) const
    {
        Vec3* objectEmissions = new Vec3[maxDepth_ - 2];
        Vec3* objectColors = new Vec3[maxDepth_ - 2];

        for (; depth<maxDepth_; depth++)
        {
            const auto hitData = getHitObjectAndDistance(ray);
            if (hitData.index_ == -1) break;

            const auto& object = objects_[hitData.index_];
            const auto intersection = ray.origin_ + ray.direction_ * hitData.distance_;
            const auto reflected = object->calculateReflections(intersection, ray.direction_, state, depth);
            ray = reflected.ray_;

            objectEmissions[depth - 2] = object->getEmission();
            objectColors[depth - 2] = object->getColor();
        }

        Vec3 pixel;
        for (int8_t i=(depth - 2); i>= 0; i--)
        {
            pixel = objectEmissions[i] + objectColors[i].mult(pixel);
        }

        delete objectEmissions;
        delete objectColors;

        return pixel;
    }

    __device__ HitData getHitObjectAndDistance(const containers::Ray& ray) const
    {
        int index = -1;
        double distance = INF;

        for (uint32_t i=0; i<objectsCount_; i++)
        {
            auto temp = objects_[i]->intersect(ray);
            if (temp && temp < distance)
            {
                distance = temp;
                index = i;
            }
        }

        return HitData(index, distance);
    }

    AObject** objects_;
    Camera camera_;
    const uint32_t height_;
    const uint32_t samples_;
    const uint32_t width_;
    const uint8_t maxDepth_;
    uint32_t objectsCount_;
};

__global__ void cudaMain(Vec3* image, ObjectData* objectsData, const uint32_t objectsCount, const uint32_t width,
    const uint32_t height, Camera camera, Vec3 vecZ, uint32_t samples, const uint8_t maxDepth)
{
    Renderer render = Renderer(samples, width, height, maxDepth, camera);
    render.setUp(objectsData, objectsCount);
    render.start(image, vecZ);
}

}  // namespace tracer::render
