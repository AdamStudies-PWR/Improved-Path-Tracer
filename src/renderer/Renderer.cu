#include "hip/hip_runtime.h"
#include <stdio.h>

#include <algorithm>

#include "containers/Vec3.hpp"
#include "objects/AObject.hpp"
#include "objects/Plane.cu"
#include "objects/Sphere.cu"
#include "scene/objects/Camera.hpp"
#include "scene/objects/ObjectData.hpp"
#include "scene/objects/RayData.hpp"
#include "utils/CudaUtils.hpp"

#include "Constants.hpp"
#include "helpers/HitData.cu"
#include "helpers/ImageData.hpp"
#include "helpers/Range.cu"
#include "helpers/RenderData.cu"


namespace tracer::renderer
{

namespace
{
using namespace containers;
using namespace scene;
using namespace scene::objects;
using namespace utils;

const float FOV_SCALE = 0.0009;
const uint16_t VIEWPORT_DISTANCE = 140;
const double INF = 1e20;

__device__ uint32_t counter = 0;

__device__ inline Range calculateRange(const uint32_t idX, const uint32_t idZ, const uint32_t width,
    const uint32_t height)
{
    auto widthAssigned = width/THREAD_SIZE;
    const auto widthOverflow = width % THREAD_SIZE;
    const auto startWidth = idX * widthAssigned + ((idX < widthOverflow) ? idX : widthOverflow);
    widthAssigned = widthAssigned + ((idX < widthOverflow) ? 1 : 0);

    auto heightAssigned = height/BLOCK_SIZE;
    const auto heightOverflow = height % BLOCK_SIZE;
    const auto startHeight = idZ * heightAssigned + ((idZ < heightOverflow) ? idZ : heightOverflow);
    heightAssigned = heightAssigned + ((idZ < heightOverflow) ? 1 : 0);

    return Range(startWidth, startHeight, (startWidth + widthAssigned), (startHeight + heightAssigned));
}

__device__ inline HitData getHitObjectAndDistance(AObject** objects, const containers::Ray& ray,
    const uint32_t objectCount)
{
    int index = -1;
    double distance = INF;

    for (uint32_t i = 0; i < objectCount; i++)
    {
        auto temp = objects[i]->intersect(ray);
        if (temp && temp < distance)
        {
            distance = temp;
            index = i;
        }
    }

    return HitData(index, distance);
}

__device__ inline Vec3 clampVector(const Vec3& vec, const double max)
{
    auto highest = (vec.xx_ > vec.yy_) ? vec.xx_ : vec.yy_;
    highest = (highest > vec.zz_) ? highest: vec.zz_;
    auto newVec = Vec3(vec.xx_/highest, vec.yy_/highest, vec.zz_/highest);
    return newVec * max;
}

__device__ inline Vec3 findLight(const RenderData& data, const AObject* lastObject, const Ray& ray,
    const Vec3& intersection)
{
    const bool isNormalNegative = (lastObject->getNormal(intersection, ray.direction_) < 0);
    double minDistance = INF;
    int32_t lightId = -1;

    Vec3 direction;
    Vec3 target;
    for (uint32_t i = 0; i < data.lightCount_; i++)
    {
        const auto extremes = data.lights_[i]->getExtremes();
        for (uint8_t j = 0; j < data.lights_[i]->getExtremesCount(); j++)
        {
            const auto distance = intersection.distance(extremes[j]);
            if (distance < minDistance)
            {
                target = extremes[j];
                direction = (target - intersection).norm();
                if ((lastObject->getNormal(intersection, direction * -1) < 0) == isNormalNegative)
                {
                    const auto propHitData =
                        getHitObjectAndDistance(data.props_, Ray(intersection, direction), data.propCount_);
                    if (propHitData.distance_ < distance && propHitData.distance_ > 0.0)
                    {
                        continue;
                    }

                    lightId = i;
                    minDistance = distance;
                    break;
                }
            }
        }
    }

    if (lightId == -1)
    {
        return Vec3();
    }

    const auto light = data.lights_[lightId];
    const auto lightAngle = light->getAngle(target, direction);
    const auto lightFactor = lightAngle/M_PI_2;
    const auto objectAngle = lastObject->getAngle(intersection, (direction * -1));
    const auto objectFactor = objectAngle/M_PI_2;
    const auto viewAngle = lastObject->getAngle(intersection, ray.direction_);
    const auto viewFactor = viewAngle/M_2_PI;
    auto factor = lightFactor * objectFactor * viewFactor;

    const auto emission = clampVector(light->getEmission(), 2.0);
    return (emission * factor) + (light->getColor()).mult(Vec3());
}

__device__ inline Vec3 deepLayers(const RenderData& data, Ray ray, uint8_t depth)
{
    Vec3* objectEmissions = new Vec3[data.maxDepth_ - 2];
    Vec3* objectColors = new Vec3[data.maxDepth_ - 2];

    for (; depth < data.maxDepth_; depth++)
    {
        const auto propHitData = getHitObjectAndDistance(data.props_, ray, data.propCount_);
        const auto lightHitData = getHitObjectAndDistance(data.lights_, ray, data.lightCount_);
        if (propHitData.index_ == -1 && lightHitData.index_ == -1) break;

        if (lightHitData.distance_ < propHitData.distance_)
        {
            const auto light = data.lights_[lightHitData.index_];
            auto scale = (1 - (0.05 * depth));
            scale = (scale >= 0.1) ? scale : 0.1;
            objectEmissions[depth - 2] = light->getEmission() * scale;
            objectColors[depth - 2] = light->getColor() * scale;
            depth++;
            break;
        }

        const auto& object = data.props_[propHitData.index_];
        const auto intersection = ray.origin_ + ray.direction_ * propHitData.distance_;
        const auto reflected = object->calculateReflections(intersection, ray.direction_, data.state_, depth);
        ray = reflected.ray_;

        objectEmissions[depth - 2] = Vec3();
        objectColors[depth - 2] = object->getColor();
    }

    Vec3 pixel = Vec3();
    for (int8_t i=(depth - 3); i>= 0; i--)
    {
        pixel = objectEmissions[i] + objectColors[i].mult(pixel);
    }

    delete objectEmissions;
    delete objectColors;

    return pixel;
}

__device__ inline Vec3 secondLayer(const RenderData& data, Ray ray, uint8_t& depth, const bool addLight)
{
    const auto propHitData = getHitObjectAndDistance(data.props_, ray, data.propCount_);
    const auto lightHitData = getHitObjectAndDistance(data.lights_, ray, data.lightCount_);
    if (propHitData.index_ == -1 && lightHitData.index_ == -1)
    {
        return Vec3();
    }

    if (lightHitData.distance_ < propHitData.distance_)
    {
        const auto light = data.lights_[lightHitData.index_];
        return (light->getEmission() + light->getColor().mult(Vec3())) * 0.95;
    }

    const auto& object = data.props_[propHitData.index_];
    const auto intersection = ray.origin_ + ray.direction_ * propHitData.distance_;
    const auto reflected = object->calculateReflections(intersection, ray.direction_, data.state_, depth);

    depth++;
    Vec3 backData;
    backData = deepLayers(data, reflected.ray_, depth) * reflected.power_;
    if (reflected.useSecond_)
    {
        backData = backData + deepLayers(data, reflected.secondRay_, depth) * reflected.secondPower_;
    }

    if (backData == Vec3() and addLight)
    {
        backData = findLight(data, object, ray, intersection) * 0.6;
    }

    return object->getColor().mult(backData);
}

__device__ inline Vec3 firstLayer(const RenderData& data, Ray ray)
{
    uint8_t depth = 0;
    const auto propHitData = getHitObjectAndDistance(data.props_, ray, data.propCount_);
    const auto lightHitData = getHitObjectAndDistance(data.lights_, ray, data.lightCount_);
    if (propHitData.index_ == -1 && lightHitData.index_ == -1)
    {
        return Vec3();
    }

    if (lightHitData.distance_ < propHitData.distance_)
    {
        const auto light = data.lights_[lightHitData.index_];
        return light->getEmission() + light->getColor().mult(Vec3());
    }

    const auto& object = data.props_[propHitData.index_];
    const auto intersection = ray.origin_ + ray.direction_ * propHitData.distance_;
    const auto reflected = object->calculateReflections(intersection, ray.direction_, data.state_, depth);

    depth++;
    Vec3 backData;
    const auto addLight = (object->getReflectionType() == Refractive or object->getReflectionType() == Specular);
    backData = secondLayer(data, reflected.ray_, depth, addLight) * reflected.power_;
    if (reflected.useSecond_)
    {
        backData = backData + secondLayer(data, reflected.secondRay_, depth, false) * reflected.secondPower_;
    }

    if (backData == Vec3() and not addLight)
    {
        backData = findLight(data, object, ray, intersection);
    }

    return object->getColor().mult(backData);
}

__device__ inline Vec3 samplePixel(const RenderData& data, const Camera* camera, ImageData* imageProperties,
    const Vec3* vecZ, const uint32_t pixelX, const uint32_t pixelZ)
{
    const auto vecX = camera->orientation_;
    const auto width = imageProperties->width_;
    const auto height = imageProperties->height_;

    auto correctionX = (width % 2 == 0) ? 0.5 : 0.0;
    auto correctionZ = (width % 2 == 0) ? 0.5 : 0.0;
    double stepX = (pixelX < width/2)
        ? width/2 - pixelX - correctionX
        : ((double)width/2 - pixelX - 1.0) + ((correctionX == 0.0) ? 1.0 : correctionX);
    double stepZ = (pixelZ < height/2)
        ? height/2 - pixelZ - correctionZ
        : ((double)height/2 - pixelZ - 1.0) + ((correctionZ == 0.0) ? 1.0 : correctionZ);

    const auto gaze = (camera->direction_ + vecX*stepX*FOV_SCALE + (*vecZ)*stepZ*FOV_SCALE).norm();

    Vec3 pixel = Vec3();
    for (uint32_t i = 0;  i < imageProperties->samples_; i++)
    {
        // Tent filter
        const auto xFactor = tent_filter(data.state_);
        const auto zFactor = tent_filter(data.state_);
        const auto tentFilter = vecX*xFactor + (*vecZ)*zFactor;
        // Tent filter

        const auto origin = camera->origin_ + vecX*stepX + (*vecZ)*stepZ + tentFilter;
        pixel = pixel + firstLayer(data, Ray(origin + camera->direction_ * VIEWPORT_DISTANCE, gaze));
    }

    pixel.xx_ = pixel.xx_/imageProperties->samples_;
    pixel.yy_ = pixel.yy_/imageProperties->samples_;
    pixel.zz_ = pixel.zz_/imageProperties->samples_;

    return pixel;
}
}  // namespace

__global__ void cudaMain(Vec3* image, AObject** props, AObject** lights, Camera* camera, Vec3* vecZ,
    ImageData* imageProperties)
{
    if (blockIdx.x == 0 and threadIdx.x == 0)
    {
        printf("\rRendering %.2f%%", (float)counter);
    }

    hiprandState state;
    auto seed = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(123456, seed, 0, &state);

    RenderData data {props, lights, imageProperties->propCount_, imageProperties->lightCount_,
        imageProperties->maxDepth_, state};

    const auto totalPixels = imageProperties->width_ * imageProperties->height_;
    const auto range = calculateRange(threadIdx.x, blockIdx.x, imageProperties->width_, imageProperties->height_);
    for (uint32_t z = range.startZ_; z < range.endZ_; z++)
    {
        for (uint32_t x = range.startX_; x < range.endX_; x++)
        {
            const auto index = z * imageProperties->width_ + x;
            image[index] = samplePixel(data, camera, imageProperties, vecZ, x, z);
            atomicAdd(&counter, 1);
        }
        printf("\rRendering %.2f%%", ((float)counter/(totalPixels)*100));
    }
}

}  // namespace tracer::render
