#include "hip/hip_runtime.h"
#include "AObject.hpp"

#include "math.h"

#include "Common.hpp"
#include "Constants.hpp"

namespace tracer::scene::objects
{

namespace
{
using namespace containers;
using namespace utils;
}  // namespace

class Sphere : public AObject
{
public:
    __device__ Sphere(double radius, Vec3 position, Vec3 emission, Vec3 color, EReflectionType reflection)
        : AObject(position, emission, color, reflection)
        , radius_(radius)
    {}

    __device__ double intersect(const Ray& ray) const override
    {
        double intersection = 0.0;

        Vec3 op = ray.origin_ - position_;
        double b = op.dot(ray.direction_);
        double delta = b*b - op.dot(op) + radius_*radius_;

        if (delta < 0) return 0;
        else delta = sqrt(delta);

        return ((intersection = -b - delta) > MARGIN)
            ? intersection : (((intersection = -b + delta) > MARGIN) ? intersection : 0.0);
        return 0.0;
    }

    __device__ RayData calculateReflections(const Vec3& intersection, const Vec3& incoming, hiprandState& state,
        const uint8_t depth) const override
    {
        const auto rawNormal = (intersection - position_).norm();
        const auto normal = incoming.dot(rawNormal) < 0 ? rawNormal * -1 : rawNormal;

        switch (reflection_)
        {
        case Specular: return handleSpecular(intersection, incoming, normal, state, depth);
        case Diffuse: return handleDiffuse(intersection, normal, state);
        case Refractive: return handleRefractive(intersection, incoming, rawNormal, normal, state, depth);
        default: printf("Uknown reflection type\n");
        }

        return {};
    }

private:
    __device__ RayData handleSpecular(const Vec3& intersection, const Vec3& incoming, const Vec3& normal,
        hiprandState& state, const uint8_t depth) const
    {
        auto specular = calculateSpecular(incoming, normal);
        auto diffuse = calculateDiffuse(normal, state);

        if (depth < 2)
        {
            if (hiprand_uniform_double(&state) > 0.9) return {Ray(intersection, diffuse), 0.08};
            else return {Ray(intersection, specular), 0.92};
        }

        if (hiprand_uniform_double(&state) > 0.9) return {Ray(intersection, diffuse), 1.0};
        else return {Ray(intersection, specular), 1.0};
    }

    __device__ RayData handleDiffuse(const Vec3& intersection, const Vec3& normal, hiprandState& state) const
    {
        auto diffuse = calculateDiffuse(normal, state);
        return {Ray(intersection, diffuse), 1.0};
    }

    __device__ RayData handleRefractive(const Vec3& intersection, const Vec3& incoming, const Vec3& rawNormal,
        const Vec3& normal, hiprandState& state, const uint8_t depth) const
    {
        auto specular = calculateSpecular(incoming, normal);

        auto refractive = calculateRefreactive(incoming, rawNormal);

        if (refractive == Vec3())
        {
            return {Ray(intersection, specular), 1.0};
        }

        if (depth < 2)
        {
            if (hiprand_uniform_double(&state) > 0.9) return {Ray(intersection, specular), 0.05};
            else return {Ray(intersection, refractive), 0.95};
        }

        if (hiprand_uniform_double(&state) > 0.95)
        {
            return {Ray(intersection, specular), 1.0};
        }
        else
        {
            return {Ray(intersection, refractive), 1.0};
        }
    }

    double radius_;
};

}  // namespace tracer::scene::objects
