#include "hip/hip_runtime.h"
#include "AObject.hpp"

#include "math.h"

#include "Constants.hpp"
#include "Helpers.hpp"


namespace tracer::scene::objects
{

namespace
{
using namespace containers;
using namespace utils;

const uint8_t PLANE_EXTREMS = 5;

__device__ double distanceToBorder(const Vec3& origin, const Vec3& border, const Vec3& impact)
{
    const auto refPoint = impact - origin;
    const auto top = border.xx_ * refPoint.xx_ + border.yy_ * refPoint.yy_ + border.zz_ * refPoint.zz_;
    const auto bottom = pow(border.xx_, 2) + pow(border.yy_, 2) + pow(border.zz_, 2);

    if (bottom == 0.0) return 0.0;

    const auto distance = top / bottom;
    return (origin + border * distance).distance(impact);
}
}  // namespace

class Plane : public AObject
{
public:
    __device__ Plane(Vec3 north, Vec3 east, Vec3 position, Vec3 emission, Vec3 color,
            EReflectionType reflection)
        : AObject(position, emission, color, reflection, PLANE_EXTREMS)
    {
        planeVector_ = (north % east).norm();

        bottomRight_ = position_ + (north * -1) + east;
        bottomLeft_ = position_ + (north * -1) + (east * -1);
        topLeft_ = position_ + north + (east * -1);
        topRight_ = position_ + north + east;

        distanceHorizontal_ = bottomLeft_.distance(bottomRight_);
        distanceVertical_ = bottomLeft_.distance(topLeft_);

        extremes_ = (Vec3*)malloc(sizeof(Vec3) * PLANE_EXTREMS);
        extremes_[0] = position_;
        extremes_[1] = position_ + north;
        extremes_[2] = position_ - north;
        extremes_[3] = position_ + east;
        extremes_[4] = position_ - east;
    }

    __device__ ~Plane()
    {
        free(extremes_);
    }

    __device__ double intersect(const Ray& ray) const override
    {
        const auto refPoint = position_ - ray.origin_;
        const auto top = planeVector_.xx_ * refPoint.xx_ + planeVector_.yy_ * refPoint.yy_
            + planeVector_.zz_ * refPoint.zz_;
        const auto bottom = planeVector_.xx_ * ray.direction_.xx_ + planeVector_.yy_ * ray.direction_.yy_
            + planeVector_.zz_ * ray.direction_.zz_;

        if (bottom == 0.0) return 0.0;

        auto distance = top/bottom;
        if (distance <= MARGIN) return 0.0;

        const auto impact = ray.origin_ + (ray.direction_ * distance);
        if (not checkIfInBounds(impact))
        {
            return 0.0;
        }

        return distance;
    }

    __device__ RayData calculateReflections(const Vec3& intersection, const Vec3& incoming,
        hiprandState& state, const uint8_t depth) const override
    {
        const auto normal = (incoming.dot(planeVector_) < 0 ? planeVector_ * -1 : planeVector_) * -1;

        switch (reflection_)
        {
        case Specular: return handleSpecular(intersection, incoming, normal, state, depth);
        case Diffuse: return handleDiffuse(intersection, normal, state);
        case Refractive: return handleRefractive(intersection, incoming, normal, normal, state, depth);
        default: printf("Uknown reflection type");
        }

        return {};
    }

    __device__ double getNormal(const Vec3&, const Vec3& incoming) const override
    {
        return incoming.dot(planeVector_);
    }

    __device__ virtual void sortExtremes(const Vec3& refPoint) const override
    {
        helpers::quickSort(extremes_, refPoint, PLANE_EXTREMS);
    }

private:
    __device__ bool checkIfInBounds(const Vec3& impact) const
    {
        auto vertical = distanceToBorder(bottomLeft_, (bottomLeft_ - bottomRight_).norm(), impact);
        if (distanceVertical_ - vertical < -MARGIN) return false;
        vertical = vertical + distanceToBorder(topLeft_, (topLeft_ - topRight_).norm(), impact);
        if (distanceVertical_  - vertical < -MARGIN or distanceVertical_  - vertical > MARGIN) return false;

        auto horizontal = distanceToBorder(bottomLeft_, (bottomLeft_ - topLeft_).norm(), impact);
        if (distanceHorizontal_ - horizontal < -MARGIN) return false;
        horizontal = horizontal + distanceToBorder(bottomRight_, (bottomRight_ - topRight_).norm(), impact);
        if (distanceHorizontal_ - horizontal < -MARGIN or distanceHorizontal_ - horizontal > MARGIN) return false;

        return true;
    }

    __device__ double getAngle(const Vec3& intersection, const Vec3& incoming) const override
    {
        const auto normal = (incoming.dot(planeVector_) < 0 ? planeVector_ * -1 : planeVector_) * -1;
        const auto cos = normal.dot(incoming) / (normal.length() * incoming.length());
        return(M_PI_2 - acos(cos));
    }


    Vec3 bottomLeft_;
    Vec3 bottomRight_;
    Vec3 planeVector_;
    Vec3 topLeft_;
    Vec3 topRight_;
    double distanceHorizontal_;
    double distanceVertical_;
};

}  // namespace tracer::scene::objects
