#include "hip/hip_runtime.h"
#include "AObject.hpp"

#include "math.h"

#include "Constants.hpp"


namespace tracer::scene::objects
{

namespace
{
using namespace containers;
using namespace utils;

struct PlaneEquastion
{
    __device__ PlaneEquastion(const double A=0.0, const double B=0.0, const double C=0.0, const double D=0.0)
        : A_(A)
        , B_(B)
        , C_(C)
        , D_(D)
    {}

    double A_;
    double B_;
    double C_;
    double D_;
};

__device__ double distanceToBorder(const Vec3& origin, const Vec3& border, const Vec3& impact)
{
    const auto refPoint = impact - origin;
    const auto top = border.xx_ * refPoint.xx_ + border.yy_ * refPoint.yy_ + border.zz_ * refPoint.zz_;
    const auto bottom = pow(border.xx_, 2) + pow(border.yy_, 2) + pow(border.zz_, 2);

    if (bottom == 0.0) return 0.0;

    const auto distance = top / bottom;
    return (origin + border * distance).distance(impact);
}
}  // namespace

class Plane : public AObject
{
public:
    __device__ Plane(Vec3 north, Vec3 east, Vec3 position, Vec3 emission, Vec3 color,
        EReflectionType reflection)
        : AObject(position, emission, color, reflection)
    {
        planeVector_ = (north % east).norm();
        const auto D = -((planeVector_.xx_ * position_.xx_) + (planeVector_.yy_ * position_.yy_)
            + (planeVector_.zz_ * position_.zz_));
        equastion_ = {planeVector_.xx_, planeVector_.yy_, planeVector_.zz_, D};

        bottomRight_ = position_ + (north * -1) + east;
        bottomLeft_ = position_ + (north * -1) + (east * -1);
        topLeft_ = position_ + north + (east * -1);
        topRight_ = position_ + north + east;

        distanceHorizontal_ = bottomLeft_.distance(bottomRight_);
        distanceVertical_ = bottomLeft_.distance(topLeft_);
    }

    __device__ double intersect(const Ray& ray) const override
    {
        const auto bottom = ray.direction_.xx_ * equastion_.A_ + ray.direction_.yy_ * equastion_.B_
            + ray.direction_.zz_ * equastion_.C_;
        if (bottom == 0.0) return 0.0;

        const auto top = -(equastion_.D_ + equastion_.A_ * ray.origin_.xx_ + equastion_.B_ * ray.origin_.yy_
            + equastion_.C_ * ray.origin_.zz_);
        auto distance = top/bottom;
        if (distance <= MARGIN) return 0.0;

        const auto impact = ray.origin_ + (ray.direction_ * distance);
        if (not checkIfInBounds(impact))
        {
            return 0.0;
        }

        return distance;
    }

    __device__ RayData calculateReflections(const Vec3& intersection, const Vec3& incoming,
        hiprandState& state, const uint8_t depth) const override
    {
        const auto normal = (incoming.dot(planeVector_) < 0 ? planeVector_ * -1 : planeVector_) * -1;

        switch (reflection_)
        {
        case Specular: return handleSpecular(intersection, incoming, normal, state, depth);
        case Diffuse: return handleDiffuse(intersection, normal, state);
        case Refractive: return handleRefractive(intersection, incoming, normal, normal, state, depth);
        default: printf("Uknown reflection type");
        }

        return {};
    }

private:
    __device__ bool checkIfInBounds(const Vec3& impact) const
    {
        auto vertical = distanceToBorder(bottomLeft_, (bottomLeft_ - bottomRight_).norm(), impact);
        if (distanceVertical_ - vertical < -MARGIN) return false;
        vertical = vertical + distanceToBorder(topLeft_, (topLeft_ - topRight_).norm(), impact);
        if (distanceVertical_  - vertical < -MARGIN or distanceVertical_  - vertical > MARGIN) return false;

        auto horizontal = distanceToBorder(bottomLeft_, (bottomLeft_ - topLeft_).norm(), impact);
        if (distanceHorizontal_ - horizontal < -MARGIN) return false;
        horizontal = horizontal + distanceToBorder(bottomRight_, (bottomRight_ - topRight_).norm(), impact);
        if (distanceHorizontal_ - horizontal < -MARGIN or distanceHorizontal_ - horizontal > MARGIN) return false;

        return true;
    }

    PlaneEquastion equastion_;
    Vec3 bottomLeft_;
    Vec3 bottomRight_;
    Vec3 planeVector_;
    Vec3 topLeft_;
    Vec3 topRight_;
    double distanceHorizontal_;
    double distanceVertical_;
};

}  // namespace tracer::scene::objects
